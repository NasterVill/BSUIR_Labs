#include "hip/hip_runtime.h"
#include ""
#include "inc/helper_image.h"

#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <iostream>
#include <iomanip>
#include <chrono>

#include "host_image_pocessing.h"
#include "device_image_pocessing.cuh"

using namespace std;

#define CUDA_DEBUG

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG) || defined(CUDA_DEBUG)
	if (result != hipSuccess)
	{
		cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << endl;
		assert(result == hipSuccess);
	}
#endif
	return result;
}

// Check errors
void postprocess(const unsigned char *in_data, const unsigned char *out_data, int size, float ms)
{
	for (int i = 0; i < size; i++)
	{
		if (in_data[i] != out_data[i])
		{
			cout << endl << "*** FAILED ***" << endl;
			cout << "Index: " << i << " exhibit: " << +in_data[i] << " result: " << +out_data[i] << endl << endl;
			break;
		}
	}
	cout << "Time spent: " << ms << endl;
}

const int NUM_REPS = 1;

int main()
{
	char file_name[] = "image.ppm";
	char cpu_resilt_file_name[] = "CPU_result.ppm";
	char gpu_resilt_file_name[] = "GPU_result.ppm";
	
	size_t width = 0;
	size_t height = 0;
	int channels = 0;

	pixel *input_data = nullptr;
	__loadPPM(
		file_name, reinterpret_cast<unsigned char **>(&input_data),
		reinterpret_cast<unsigned int*>(&width), 
		reinterpret_cast<unsigned int*>(&height),
		reinterpret_cast<unsigned int*>(&channels)
	);

	cout << width << " " << height << " " << channels << endl << endl;

	const size_t padded_width = width + 2;
	const size_t padded_height = height + 2;

	const size_t size = width * height;

	const size_t width_in_bytes = width * sizeof(pixel);
	const size_t padded_width_in_bytes = padded_width * sizeof(pixel);

	const size_t size_in_bytes = width_in_bytes * height;

	pixel *cpu_output_data = new pixel[size];
	pixel *gpu_output_data = new pixel[size];

	// ********************************************************************************************************

	cout << "Filtering via CPU" << endl;
	auto start_cpu = chrono::steady_clock::now();
	for (int i = 0; i < NUM_REPS; i++)
	{
		ApplySquareAverageFilter(input_data, cpu_output_data, width, height);
	}
	auto end_cpu = chrono::steady_clock::now();
	auto cpu_time = end_cpu - start_cpu;
	cout << "CPU time: " << chrono::duration<double, milli>(cpu_time).count() / NUM_REPS << endl;

	// ********************************************************************************************************

	// ********************************************************************************************************

	size_t input_pitch = 0;
	pixel *padded_input = PadDataByOnePixel(input_data, width, height);
	unsigned char *pitched_input_data = nullptr;
	checkCuda(hipMallocPitch(reinterpret_cast<void **>(&pitched_input_data), &input_pitch, padded_width_in_bytes, padded_height));
	checkCuda(hipMemcpy2D(
		pitched_input_data,
		input_pitch,
		reinterpret_cast<unsigned char **>(padded_input),
		padded_width_in_bytes,
		padded_width_in_bytes,
		padded_height,
		hipMemcpyHostToDevice
	));

	size_t output_pitch = 0;
	unsigned char *pitched_output_data = nullptr;
	hipMallocPitch(reinterpret_cast<void**>(&pitched_output_data), &output_pitch, width_in_bytes, height);

	float time = 0;
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));

	//
	cout << "Filtering via GPU" << " pitch: " << input_pitch << endl;

	int aligned_width = (input_pitch + AMOUNT_OF_THREADS_X - 1) / AMOUNT_OF_THREADS_X;
	int aligned_height = (padded_height + AMOUNT_OF_THREADS_Y - 1) / AMOUNT_OF_THREADS_Y;
	dim3 dimGrid(aligned_width, aligned_height, 1);
	dim3 dimBlock(AMOUNT_OF_THREADS_X, AMOUNT_OF_THREADS_Y, 1);
	
	// warm up
	ApplySquareAverageFilter <<<dimGrid, dimBlock>>> (
			pitched_input_data, pitched_output_data,
			width_in_bytes, height,
			padded_width_in_bytes, padded_height,
			input_pitch, output_pitch
		);

	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
	{
		ApplySquareAverageFilter <<<dimGrid, dimBlock>>> (
				pitched_input_data, pitched_output_data,
				width_in_bytes, height,
				padded_width_in_bytes, padded_height,
				input_pitch, output_pitch
			);
	}
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));
	time /= NUM_REPS;

	cout << "GPU time: " << time << endl;

	checkCuda(hipMemcpy2D(
		reinterpret_cast<unsigned char *>(gpu_output_data),
		width_in_bytes,
		pitched_output_data,
		output_pitch,
		width_in_bytes,
		height,
		hipMemcpyDeviceToHost
	));

	// ********************************************************************************************************

	// check
	postprocess(reinterpret_cast<unsigned char *>(cpu_output_data), reinterpret_cast<unsigned char *>(gpu_output_data), size, time);

	__savePPM(cpu_resilt_file_name, reinterpret_cast<unsigned char *>(cpu_output_data), width, height, channels);
	__savePPM(gpu_resilt_file_name, reinterpret_cast<unsigned char *>(gpu_output_data), width, height, channels);
	
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	checkCuda(hipFree(pitched_input_data));
	checkCuda(hipFree(pitched_output_data));
	delete[] input_data;
	delete[] cpu_output_data;
	delete[] gpu_output_data;
}