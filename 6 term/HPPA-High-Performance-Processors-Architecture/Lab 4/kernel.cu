#include "hip/hip_runtime.h"
#include ""
#include "host_helper.h"
#include "device_helper.cuh"

#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <iostream>
#include <iomanip>
#include <chrono>

using namespace std;

#define CUDA_DEBUG

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG) || defined(CUDA_DEBUG)
	if (result != hipSuccess)
	{
		cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << endl;
		assert(result == hipSuccess);
	}
#endif
	return result;
}

// Check errors
void postprocess(const short *in_data, const short *out_data, int size, float ms)
{
	for (int i = 0; i < size; i++)
	{
		if (in_data[i] != out_data[i])
		{
			cout << endl << "*** FAILED ***" << endl;
			cout << "Index: " << i << " exhibit: " << +in_data[i] << " result: " << +out_data[i] << endl << endl;
			break;
		}
	}
	printf("Time spent: %f\n" ,ms);
}

int main()
{
	const int width = 1222;
	const int height = 144;
	const int size = width * height * sizeof(short);

	int aligned_width = (width + MATRIX_TILE - 1) / MATRIX_TILE * (height + MATRIX_TILE - 1) / MATRIX_TILE;
	dim3 dimGrid(aligned_width, 1, 1);
	dim3 dimBlock(MATRIX_TILE * MATRIX_TILE, 1, 1);

	short *host_input_data = (short*)malloc(size);
	short *device_to_host_memory = (short*)malloc(size);
	short *cpu_rotated_data = (short*)malloc(size);

	short *device_input_data, *device_rotated_data;
	checkCuda(hipMalloc(&device_input_data, size));
	checkCuda(hipMalloc(&device_rotated_data, size));

	// host
	for (int j = 0; j < height; j++)
	{
		for (int i = 0; i < width; i++)
		{ 
			host_input_data[j * width + i] = j * width + i;
		}
	}

	// ********************************************************************************************************

	cout << "Rotating via CPU" << endl;
	auto start_cpu = chrono::steady_clock::now();
	for (int i = 0; i < NUM_REPS; i++)
	{
		MirrotAndRotateMatrix(host_input_data, cpu_rotated_data, width, height);
	}
	auto end_cpu = chrono::steady_clock::now();
	auto cpu_time = end_cpu - start_cpu;
	cout << "CPU time: " << chrono::duration<double, milli>(cpu_time).count() << endl;

	checkCuda(hipMemcpy(device_input_data, host_input_data, size, hipMemcpyHostToDevice));

	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	cout << "Rotating via GPU" << endl;
	checkCuda(hipMemset(device_rotated_data, 0, size));
	memset(device_to_host_memory, 0, size);

	__gpu_RotateMatrix <<<dimGrid, dimBlock>>> (device_input_data, device_rotated_data, width, height);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
	{
		__gpu_RotateMatrix <<<dimGrid, dimBlock>>> (device_input_data, device_rotated_data, width, height);
	}
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(device_to_host_memory, device_rotated_data, size, hipMemcpyDeviceToHost));
	postprocess(cpu_rotated_data, device_to_host_memory, width * height, ms);

	/*for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			cout << setw(5) << device_to_host_memory[i * width + j];
			if (j == width - 1) cout << endl;
		}
	}

	cout << endl << endl;

	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			cout << setw(5) << cpu_rotated_data[i * width + j];
			if (j == width - 1) cout << endl;
		}
	}*/

	// *******************************************************************************************************

	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	checkCuda(hipFree(device_rotated_data));
	checkCuda(hipFree(device_input_data));
	free(host_input_data);
	free(device_to_host_memory);
	free(cpu_rotated_data);
}