#include "hip/hip_runtime.h"
#include "device_helper.cuh"
#include "host_helper.h"

__global__ void __gpu_MirrorMatrix(short* input_data, short* output_data, int width, int height)
{
	__shared__ ushort2 tile[MATRIX_TILE * MATRIX_TILE];

	// x and y for int 'array' (which may be a bit jagged ('cause we can have 1 excessive short in the end of an array))
	int x = blockIdx.x * MATRIX_TILE * MATRIX_TILE + threadIdx.x;
	int max_x = (width * height + 1) / 2;

	if (x >= max_x)
	{
		return;
	}

	ushort2* interpreted_input = (ushort2*)input_data;

	int new_y = x * 2 / width;
	int new_x = x * 2 % width;

	if (x * 2 + 1 < width * height)
	{
		tile[threadIdx.x] = interpreted_input[x];
	}
	else
	{
		// process last (odd) element, if needed
		tile[threadIdx.x] = interpreted_input[max_x - 1];
		output_data[MLPIM(new_x, new_y, width)] = tile[threadIdx.x].x;
		return;
	}

	ushort2 res = tile[threadIdx.x];
	output_data[MLPIM(new_x, new_y, width)] = res.x;

	/*printf("x: %d, new_x:%d, new_y:%d, rex.x:%d, res.y:%d, MLPIM.y: %d, MLPIM.y :%d, width:%d, cond:%d  %d\n",
		x, new_x, new_y, res.x, res.y,
		MLPIM(new_x, new_y, width), (new_x == width - 1) ? MLPIM(0, new_y + 1, width) : MLPIM((new_x + 1), new_y, width),
		width, (new_x == width - 1), ((new_y + 1) * width - new_x - 1));*/

	if (new_x == width - 1)
	{
		output_data[MLPIM(0, (new_y + 1), width)] = res.y;
	}
	else
	{
		output_data[MLPIM((new_x + 1), new_y, width)] = res.y;
	}
}

__global__ void __gpu_RotateMatrix(short* input_data, short* output_data, int width, int height)
{
	__shared__ ushort2 tile[MATRIX_TILE * MATRIX_TILE];

	// x and y for int 'array' (which may be a bit jagged ('cause we can have 1 excessive short in the end of an array))
	int x = blockIdx.x * MATRIX_TILE * MATRIX_TILE + threadIdx.x;
	int size = width * height;
	int max_x = (size + 1) / 2;

	if (x >= max_x)
	{
		return;
	}

	ushort2* interpreted_input = (ushort2*)input_data;

	int new_y = x * 2 / width;
	int new_x = x * 2 % width;

	if (x * 2 + 1 < size)
	{
		tile[threadIdx.x] = interpreted_input[x];
	}
	else
	{
		// process last (odd) element, if needed
		tile[threadIdx.x].x = interpreted_input[max_x - 1].x;
		output_data[size - 1 - LPIM(new_y, new_x, height)] = tile[threadIdx.x].x;
		return;
	}

	ushort2 res = tile[threadIdx.x];
	output_data[size - 1 - LPIM(new_y, new_x, height)] = res.x;

	if (new_x == width - 1)
	{
		output_data[size - 1 - LPIM((new_y + 1), 0, height)] = res.y;
	}
	else
	{
		output_data[size - 1 - LPIM(new_y, (new_x + 1), height)] = res.y;
	}
}