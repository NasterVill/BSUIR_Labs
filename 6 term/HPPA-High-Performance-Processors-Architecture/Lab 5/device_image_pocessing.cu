#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "device_image_pocessing.cuh"

#include <stdio.h>

__global__ void ApplySquareAverageFilter(
	unsigned char *input_data,
	unsigned char *output_data,
	const int width,
	const int height,
	const int padded_width,
	const int padded_height,
	const int in_pitch,
	const int out_pitch
)
{
	const int x = blockIdx.x * AMOUNT_OF_THREADS_X + threadIdx.x;
	const int y = blockIdx.y * AMOUNT_OF_THREADS_Y + threadIdx.y;

	const int int_widht = in_pitch / sizeof(int);
	const int width_border = (width + sizeof(int) - 1) / sizeof(int);

	uchar4 *reintterpreted_input = reinterpret_cast<uchar4*>(input_data);
	uchar4 *reintterpreted_output = reinterpret_cast<uchar4*>(output_data);

	// padding memory so that we could read a bit more than int per thread
	__shared__ uchar4 shared_memory[AMOUNT_OF_THREADS_Y + 2][AMOUNT_OF_THREADS_X + 2];

	if (x < int_widht || y < padded_height)
	{
		uchar4 *reintterpreted_input = reinterpret_cast<uchar4*>(input_data);
		uchar4 *reintterpreted_output = reinterpret_cast<uchar4*>(output_data);

		// initialized shared memory (first part)
		shared_memory[threadIdx.y][threadIdx.x] = reintterpreted_input[y * int_widht + x];

		if (threadIdx.y < 2)
		{
			// copying 2 additional bottom rows (beyond AMOUNT_OF_THREADS_Y - 1)
			if (y + AMOUNT_OF_THREADS_Y <= height)
			{
				shared_memory[AMOUNT_OF_THREADS_Y + threadIdx.y][threadIdx.x] = reintterpreted_input[(AMOUNT_OF_THREADS_Y + y) * int_widht + x];
			}
			if (threadIdx.x < AMOUNT_OF_THREADS_Y + 2)
			{
				int temp_x = blockIdx.x * AMOUNT_OF_THREADS_X + threadIdx.y;
				int temp_y = blockIdx.y * AMOUNT_OF_THREADS_Y + threadIdx.x;

				if (temp_x < int_widht && temp_y < padded_height)
				{
					shared_memory[threadIdx.x][AMOUNT_OF_THREADS_X + threadIdx.y] = reintterpreted_input[temp_y * int_widht + AMOUNT_OF_THREADS_X + temp_x];
				}
			}
		}
	}

	__syncthreads();

	if (y >= height || x >= width_border)
	{
		return;
	}

	// till now every thread was assigned to a specific int,
	// but now 1 thread will work with 6 ints , so we could form a 
	// transaction to a global memory

	uchar4 generated_int = { 0 };

	uchar4 first_int = shared_memory[threadIdx.y][threadIdx.x];
	uchar4 second_int = shared_memory[threadIdx.y][threadIdx.x + 1];
	uchar4 third_int = shared_memory[threadIdx.y + 1][threadIdx.x];
	uchar4 fourth_int = shared_memory[threadIdx.y + 1][threadIdx.x + 1];
	uchar4 fifth_int = shared_memory[threadIdx.y + 2][threadIdx.x];
	uchar4 sixth_int = shared_memory[threadIdx.y + 2][threadIdx.x + 1];

	generated_int.x = (first_int.x + first_int.y + first_int.z +
					third_int.x + third_int.y + third_int.z +
					fifth_int.x + fifth_int.y + fifth_int.z) / 9;

	generated_int.y = (first_int.y + first_int.z + first_int.w +
					third_int.y + third_int.z + third_int.w +
					fifth_int.y + fifth_int.z + fifth_int.w) / 9;

	generated_int.z = (first_int.z + first_int.w + second_int.x +
					third_int.z + third_int.w + fourth_int.x +
					fifth_int.z + fifth_int.w + sixth_int.x) / 9;

	generated_int.w = (first_int.w + second_int.x + second_int.y +
					third_int.w + fourth_int.x + fourth_int.y +
					fifth_int.w + sixth_int.x + sixth_int.y) / 9;

	const int output_int_width = out_pitch / sizeof(int);

	reintterpreted_output[y * output_int_width + x] = generated_int;
}