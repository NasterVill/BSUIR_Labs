#include "hip/hip_runtime.h"
#include ""
#include "inc/helper_image.h"

#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <iostream>
#include <iomanip>
#include <chrono>

#include "host_image_pocessing.h"
#include "device_image_pocessing.cuh"

using namespace std;

#define CUDA_DEBUG

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG) || defined(CUDA_DEBUG)
	if (result != hipSuccess)
	{
		cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << endl;
		assert(result == hipSuccess);
	}
#endif
	return result;
}

// Check errors
void postprocess(const unsigned char *in_data, const unsigned char *out_data, int size, float ms)
{
	for (int i = 0; i < size; i++)
	{
		if (in_data[i] != out_data[i])
		{
			cout << endl << "*** FAILED ***" << endl;
			cout << "Index: " << i << " exhibit: " << +in_data[i] << " result: " << +out_data[i] << endl << endl;
			break;
		}
	}
	cout << "Time spent: " << ms << endl;
}

const int NUM_REPS = 1;

int main()
{
	char file_name[] = "Creedence.pgm";
	// char file_name[] = "city.pgm";
	// char file_name[] = "ray.pgm";
	// char file_name[] = "CITY_LARGE.pgm";
	char cpu_resilt_file_name[] = "CPU_result.pgm";
	char gpu_resilt_file_name[] = "GPU_result.pgm";
	
	size_t width = 0;
	size_t height = 0;
	int channels = 0;

	unsigned char *input_data = nullptr;
	__loadPPM(
		file_name, &input_data,
		reinterpret_cast<unsigned int*>(&width), 
		reinterpret_cast<unsigned int*>(&height),
		reinterpret_cast<unsigned int*>(&channels)
	);

	cout << width << " " << height << " " << channels << endl << endl;

	const size_t padded_width = width + 2;
	const size_t padded_height = height + 2;

	const size_t size = width * height;
	const size_t padded_size = width * height;

	unsigned char *cpu_output_data = new unsigned char[size];
	unsigned char *gpu_output_data = new unsigned char[size];

	// ********************************************************************************************************

	cout << "Filtering via CPU" << endl;
	auto start_cpu = chrono::steady_clock::now();
	for (int i = 0; i < NUM_REPS; i++)
	{
		ApplySquareAverageFilter(input_data, cpu_output_data, width, height);
	}
	auto end_cpu = chrono::steady_clock::now();
	auto cpu_time = end_cpu - start_cpu;
	cout << "CPU time: " << chrono::duration<double, milli>(cpu_time).count() / NUM_REPS << endl;

	// ********************************************************************************************************

	// ********************************************************************************************************

	size_t input_pitch = 0;
	unsigned char *padded_input = PadDataByOneByte(input_data, width, height);
	unsigned char *pitched_input_data = nullptr;
	checkCuda(hipMallocPitch(reinterpret_cast<void**>(&pitched_input_data), &input_pitch, padded_width, padded_height));
	checkCuda(hipMemcpy2D(
		pitched_input_data,
		input_pitch,
		padded_input,
		padded_width,
		padded_width,
		padded_height,
		hipMemcpyHostToDevice
	));

	size_t output_pitch = 0;
	unsigned char *pitched_output_data = nullptr;
	hipMallocPitch(reinterpret_cast<void**>(&pitched_output_data), &output_pitch, width, height);

	float time = 0;
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));

	//
	cout << "Filtering via GPU" << " pitch: " << input_pitch << endl;

	int aligned_width = (input_pitch + AMOUNT_OF_THREADS_X - 1) / AMOUNT_OF_THREADS_X;
	int aligned_height = (height + AMOUNT_OF_THREADS_Y - 1) / AMOUNT_OF_THREADS_Y;
	dim3 dimGrid(aligned_width, aligned_height, 1);
	dim3 dimBlock(AMOUNT_OF_THREADS_X, AMOUNT_OF_THREADS_Y, 1);
	
	// warm up
	ApplySquareAverageFilter << <dimGrid, dimBlock >> > (
		pitched_input_data, pitched_output_data,
		width, height,
		padded_width, padded_height,
		input_pitch, output_pitch
		);

	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
	{
		ApplySquareAverageFilter <<<dimGrid, dimBlock>>> (
				pitched_input_data, pitched_output_data,
				width, height,
				padded_width, padded_height,
				input_pitch, output_pitch
			);
	}
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));
	time /= NUM_REPS;

	cout << "GPU time: " << time << endl;

	checkCuda(hipMemcpy2D(
		gpu_output_data,
		width,
		pitched_output_data,
		output_pitch,
		width,
		height,
		hipMemcpyDeviceToHost
	));

	// ********************************************************************************************************

	// check
	postprocess(cpu_output_data, gpu_output_data, size, time);

	__savePPM(cpu_resilt_file_name, cpu_output_data, width, height, channels);
	__savePPM(gpu_resilt_file_name, gpu_output_data, width, height, channels);
	
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	checkCuda(hipFree(pitched_input_data));
	checkCuda(hipFree(pitched_output_data));
	delete[] input_data;
	delete[] cpu_output_data;
	delete[] gpu_output_data;
}